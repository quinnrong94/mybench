#include "hip/hip_runtime.h"
#include "common.h"

#include "hip/hip_fp16.h"

#define LOOP 10000

__global__
void fmla(half2 *in, int count) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < count) {
        half2 val = in[idx];
        for (int i = 0; i < LOOP; ++i) {
            val += val * val;
            val += val * val;
            val += val * val;
            val += val * val;

            val += val * val;
            val += val * val;
            val += val * val;
            val += val * val;

            val += val * val;
            val += val * val;
            val += val * val;
            val += val * val;

            val += val * val;
            val += val * val;
            val += val * val;
            val += val * val;
        }
        in[idx] = val;
    }
}

int main() {
    int count = 1<<20;
    auto h_in = getInitData<half>(count);
    auto in = getDeviceArray(count, h_in.get());
    printDeviceArray(in.get(), 10);

    int THREADS_PER_BLOCK = 256;
    fmla<<<BLOCKS_PER_GRID(count / 2), THREADS_PER_BLOCK>>>((half2*)in.get(), count);
    printDeviceArray(in.get(), 10);

    {
        int wc = 5;
        for (int i = 0; i < wc; ++i) {
            fmla<<<BLOCKS_PER_GRID(count / 2), THREADS_PER_BLOCK>>>((half2*)in.get(), count);
        }
        CK_CUDA_THROW(hipDeviceSynchronize());

        Timer timer;
        timer.start("fmla");
        int ic = 10;
        for (int i = 0; i < ic; ++i) {
            fmla<<<BLOCKS_PER_GRID(count / 2), THREADS_PER_BLOCK>>>((half2*)in.get(), count);
        }
        CK_CUDA_THROW(hipDeviceSynchronize());
        float ms = timer.stop(ic);
        printf("GFlops = %f\n", 1.0 * count * 16 * 2 * LOOP / ms / 1000000.0);
    }

    return 0;
}
